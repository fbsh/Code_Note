#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "Error.h"

/*
**Execution Config 1
* N = 4
**Execution Config 2
* N = 16
**Execution Config 3
* N = 32
**Execution Config 4
* N = 64
*/

#define N 64

__global__ void additionMatricesKernel(int* d_a, int* d_b, int* d_c) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    d_c[i * N + j] = d_a[i * N + j] + d_b[i * N + j];
}

void onDevice(int h_a[][N], int h_b[][N], int h_c[][N]) {
    // declare GPU memory pointers
    int *d_a, *d_b, *d_c;

    const int ARRAY_BYTES = N * N * sizeof(int);

    // allocate  memory on the GPU
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_b, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_c, ARRAY_BYTES));

    // copy data from CPU the GPU
    HANDLER_ERROR_ERR(
        hipMemcpy(d_a, h_a, ARRAY_BYTES, hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(
        hipMemcpy(d_b, h_b, ARRAY_BYTES, hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(
        hipMemcpy(d_c, h_c, ARRAY_BYTES, hipMemcpyHostToDevice));

    // execution configuration
    dim3 GridBlocks(N / 2, N / 2);
    dim3 ThreadsBlocks(N / 2, N / 2);

    // run the kernel
    additionMatricesKernel<<<GridBlocks, ThreadsBlocks>>>(d_a, d_b, d_c);
    HANDLER_ERROR_MSG("kernel panic!!!");

    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(
        hipMemcpy(h_c, d_c, ARRAY_BYTES, hipMemcpyDeviceToHost));

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_a));
    HANDLER_ERROR_ERR(hipFree(d_b));
    HANDLER_ERROR_ERR(hipFree(d_c));
}

void test(int h_a[][N], int h_b[][N], int h_c[][N]) {
    int i, j;
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            assert(h_a[i][j] + h_b[i][j] == h_c[i][j]);
        }
    }

    printf("-: successful execution :-\n");
}

void onHost() {
    int i, j;
    int h_a[N][N], h_b[N][N], h_c[N][N];

    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            h_a[i][j] = h_b[i][j] = i + j;
            h_c[i][j] = 0;
        }
    }

    // call device configuration
    onDevice(h_a, h_b, h_c);
    test(h_a, h_b, h_c);
}

int main() {
    onHost();
}
