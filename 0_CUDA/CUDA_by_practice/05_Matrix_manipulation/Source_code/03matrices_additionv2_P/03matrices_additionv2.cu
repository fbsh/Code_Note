#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "Error.h"

#define N 500

__global__ void additionMatricesKernel(int* d_a, int* d_b, int* d_c) {
    // -:YOUR CODE HERE:-
}

void onDevice(int h_a[][N], int h_b[][N], int h_c[][N]) {
    // declare GPU memory pointers
    int *d_a, *d_b, *d_c;

    const int ARRAY_BYTES = N * N * sizeof(int);

    // allocate  memory on the GPU
    // -:YOUR CODE HERE:-

    // copy data from CPU the GPU
    // -:YOUR CODE HERE:-

    // execution configuration
    dim3 GridBlocks(4, 4);
    dim3 ThreadsBlocks(8, 8);

    // run the kernel
    additionMatricesKernel<<<GridBlocks, ThreadsBlocks>>>(d_a, d_b, d_c);
    HANDLER_ERROR_MSG("kernel panic!!!");

    // copy data back from the GPU to the CPU
    // -:YOUR CODE HERE:-

    // free GPU memory
    // -:YOUR CODE HERE:-
}

void test(int h_a[][N], int h_b[][N], int h_c[][N]) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            assert(h_a[i][j] + h_b[i][j] == h_c[i][j]);
        }
    }

    printf("-: successful execution :-\n");
}

void onHost() {
    int i, j;
    int h_a[N][N], h_b[N][N], h_c[N][N];

    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            h_a[i][j] = h_b[i][j] = i + j;
            h_c[i][j] = 0;
        }
    }

    // call device configuration
    onDevice(h_a, h_b, h_c);
    test(h_a, h_b, h_c);
}

int main() {
    onHost();
}
