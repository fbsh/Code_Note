#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Error.h"
#include "GpuTimer.h"
#include "Matrix.h"

#define N 4

__global__ void matrixMultiplicationKernel(Matrix<float> d_a,
                                           Matrix<float> d_b,
                                           Matrix<float> d_c) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int k;
    float tmp;
    float a, b;

    while (i < N) {
        j = threadIdx.y + blockDim.y * blockIdx.y;
        while (j < N) {
            tmp = 0.0;
            a = 0.0;
            b = 0.0;
            for (k = 0; k < N; k++) {
                a = d_a.getElement(i, k);
                b = d_b.getElement(k, j);
                tmp += a * b;
            }
            d_c.setElement(i, j, tmp);
            j += blockDim.y * gridDim.y;
        }
        i += blockDim.x * gridDim.x;
    }
}

void onDevice(Matrix<float> h_a, Matrix<float> h_b, Matrix<float> h_c) {
    // declare GPU data
    Matrix<float> d_a, d_b, d_c;
    d_a.width = h_a.width;
    d_a.height = h_a.height;

    d_b.width = h_b.width;
    d_b.height = h_b.height;

    d_c.width = h_c.width;
    d_c.height = h_c.height;

    // start timer
    GpuTimer timer;
    timer.Start();

    const int ARRAY_BYTES = N * N * sizeof(float);

    // allocate  memory on the GPU
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_b.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_c.elements, ARRAY_BYTES));

    // copy data from CPU the GPU
    HANDLER_ERROR_ERR(hipMemcpy(d_a.elements, h_a.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(hipMemcpy(d_b.elements, h_b.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));

    // execution configuration
    dim3 GridBlocks(2, 2);
    dim3 ThreadsBlocks(2, 2);

    // run the kernel
    matrixMultiplicationKernel<<<GridBlocks, ThreadsBlocks>>>(d_a, d_b, d_c);
    HANDLER_ERROR_MSG("kernel panic!!!");

    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_c.elements, d_c.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));

    // stop timer
    timer.Stop();

    // print time
    printf("Time :  %f ms\n", timer.Elapsed());

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_a.elements));
    HANDLER_ERROR_ERR(hipFree(d_b.elements));
    HANDLER_ERROR_ERR(hipFree(d_c.elements));
}

void test() {
    Matrix<float> h_a, h_b, h_c;

    h_a.width = N;
    h_a.height = N;

    h_b.width = N;
    h_b.height = N;

    h_c.width = N;
    h_c.height = N;

    h_a.elements = (float*)malloc(h_a.width * h_b.height * sizeof(int));
    h_b.elements = (float*)malloc(h_b.width * h_b.height * sizeof(int));
    h_c.elements = (float*)malloc(h_c.width * h_c.height * sizeof(int));

    int i, j, k = 1;

    for (i = 0; i < h_a.height; i++) {
        for (j = 0; j < h_a.height; j++) {
            h_a.setElement(i, j, k);
            h_b.setElement(i, j, 1.0);
            h_c.setElement(i, j, 0.0);
            k++;
        }
    }

    // call device configuration
    onDevice(h_a, h_b, h_c);

    // print  result
    for (i = 0; i < h_c.width; i++) {
        for (j = 0; j < h_c.height; j++) {
            printf("%.2f ", h_c.elements[i * h_c.width + j]);
        }
        printf("\n");
    }

    printf("-: successful execution :-\n");

    free(h_a.elements);
    free(h_b.elements);
    free(h_c.elements);
}

int main() {
    test();
}
