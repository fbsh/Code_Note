#include "hip/hip_runtime.h"
#include <assert.h>
#include "Error.h"
#include "GpuTimer.h"
#include "Vector.h"

const int BLOCKSIZE = 128;
const int NUMBLOCKS = 1000;
const int N = BLOCKSIZE * NUMBLOCKS;
const int ARRAY_BYTES = N * sizeof(int);

__global__ void tileKernelv2(Vector<int> d_in, Vector<int> d_out) {
    int x = threadIdx.x;
    int i = x + blockIdx.x * blockDim.x;

    __shared__ int As[BLOCKSIZE][5];
    int aux;

    As[x][0] = d_in.getElement(i - 2);
    As[x][1] = d_in.getElement(i - 1);
    As[x][2] = d_in.getElement(i);
    As[x][3] = d_in.getElement(i + 1);
    As[x][4] = d_in.getElement(i + 2);

    __syncthreads();

    aux = (As[x][0] + As[x][1] + As[x][2] + As[x][3] + As[x][4]) / 5.0f;

    d_out.setElement(i, aux);
}

void onDevice(Vector<int> h_in, Vector<int> h_out) {
    Vector<int> d_in, d_out;
    d_in.length = N;
    d_out.length = N;

    // allocate  memory on the GPU
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_in.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_out.elements, ARRAY_BYTES));

    // copy data from CPU the GPU
    HANDLER_ERROR_ERR(hipMemcpy(d_in.elements, h_in.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));

    // launch kernel
    tileKernelv2<<<N / BLOCKSIZE, BLOCKSIZE>>>(d_in, d_out);
    HANDLER_ERROR_MSG("kernel panic!!!");

    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_out.elements, d_out.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_in.elements));
    HANDLER_ERROR_ERR(hipFree(d_out.elements));
}

void test(Vector<int> h_in, Vector<int> h_out) {
    int aux = 0;
    for (int i = 2; i < N - 2; i++) {
        aux = (h_in.getElement(i - 2) + h_in.getElement(i - 1) +
               h_in.getElement(i) + h_in.getElement(i + 1) +
               h_in.getElement(i + 2)) /
              5.0f;
        assert(aux == h_out.getElement(i));
    }
}

void onHost() {
    Vector<int> h_in, h_out;
    h_in.length = N;
    h_out.length = N;

    h_in.elements = (int*)malloc(ARRAY_BYTES);
    h_out.elements = (int*)malloc(ARRAY_BYTES);

    for (int i = 0; i < N; i++) {
        h_in.setElement(i, 2 * i);
    }

    // call device configuration
    onDevice(h_in, h_out);

    // testing
    test(h_in, h_out);

    printf("-: successful execution :-\n");

    free(h_in.elements);
    free(h_out.elements);
}

int main(void) {
    onHost();
    return 0;
}