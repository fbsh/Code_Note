/*
Source: http://docs.nvidia.com/cuda/hiprand/index.html#topic_1_2_2
*/

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#include "common/Error.h"

int randomNumbersGenerator(int n) {
    size_t i;
    hiprandGenerator_t gen;
    float *d_data, *h_data;

    const int ARRAY_BYTES = n * sizeof(float);

    /* Allocate n floats on host */
    h_data = (float*)calloc(n, sizeof(float));

    /* Allocate n floats on device */
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_data, ARRAY_BYTES));

    /* Create pseudo-random number generator */
    HANDLER_CURAND(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

    /* Set seed 1234ULL = unsigned long long */
    srand48(time(NULL));
    HANDLER_CURAND(hiprandSetPseudoRandomGeneratorSeed(gen, lrand48()));

    /* Set seed 1234ULL = unsigned long long
    Use this to generate the same random numbers*/
    // HANDLER_CURAND(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    /* Generate n floats on device */
    HANDLER_CURAND(hiprandGenerateUniform(gen, d_data, n));

    /* Copy device memory to host */
    HANDLER_ERROR_ERR(
        hipMemcpy(h_data, d_data, ARRAY_BYTES, hipMemcpyDeviceToHost));

    /* Show result */
    for (i = 0; i < n; i++) {
        printf("%1.4f ", h_data[i]);
    }
    printf("\n");

    /* Cleanup */
    HANDLER_CURAND(hiprandDestroyGenerator(gen));
    HANDLER_ERROR_ERR(hipFree(d_data));
    free(h_data);
    return EXIT_SUCCESS;
}

int main(int argc, char* argv[]) {
    randomNumbersGenerator(100);
}