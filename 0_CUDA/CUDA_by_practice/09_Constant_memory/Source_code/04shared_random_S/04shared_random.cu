#include "hip/hip_runtime.h"
#include <hiprand.h>
#include "common/Error.h"
#include "common/GpuTimer.h"
#include "common/Vector.h"

#define N (32 * 1024)
#define THREADS 256
#define BLOCKS 32

const int ARRAY_BYTES = N * sizeof(float);
const int P_ARRAY_BYTES = BLOCKS * sizeof(float);

__global__ void dotKernel(Vector<float> d_a,
                          Vector<float> d_b,
                          Vector<float> d_c) {
    __shared__ float cache[THREADS];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < N) {
        temp += d_a.getElement(tid) * d_b.getElement(tid);
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cacheIndex] = temp;

    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x / 2;
    while (i != 0) {
        __syncthreads();
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        i /= 2;
    }

    if (cacheIndex == 0)
        d_c.setElement(blockIdx.x, cache[0]);
}

int randomNumbersGenerator(Vector<float> d_data, int n) {
    hiprandGenerator_t gen;

    /* Create pseudo-random number generator */
    HANDLER_CURAND(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

    /* Set seed 1234ULL = unsigned long long */
    srand48(time(NULL));
    HANDLER_CURAND(hiprandSetPseudoRandomGeneratorSeed(gen, lrand48()));

    /* Set seed 1234ULL = unsigned long long
    Use this to generate the same random numbers*/
    // HANDLER_CURAND(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

    /* Generate n floats on device */
    HANDLER_CURAND(hiprandGenerateUniform(gen, d_data.elements, n));

    /* Cleanup */
    HANDLER_CURAND(hiprandDestroyGenerator(gen));

    return EXIT_SUCCESS;
}

void onDevice(Vector<float> h_c) {
    Vector<float> d_a, d_b, d_c;
    d_a.length = N;
    d_b.length = N;
    d_c.length = BLOCKS;

    // start timer
    GpuTimer timer;
    timer.Start();

    // allocate  memory on the GPU
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_b.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_c.elements, P_ARRAY_BYTES));

    randomNumbersGenerator(d_a, N);
    randomNumbersGenerator(d_b, N);

    dotKernel<<<BLOCKS, THREADS>>>(d_a, d_b, d_c);
    HANDLER_ERROR_MSG("kernel panic!!!");

    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_c.elements, d_c.elements, P_ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));

    // stop timer
    timer.Stop();

    // print time
    printf("Time :  %f ms\n", timer.Elapsed());

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_a.elements));
    HANDLER_ERROR_ERR(hipFree(d_b.elements));
    HANDLER_ERROR_ERR(hipFree(d_c.elements));
}

void test() {
    Vector<float> h_c;

    // declare vectore to store results
    h_c.length = BLOCKS;

    h_c.elements = (float*)malloc(P_ARRAY_BYTES);

    // call device configuration
    onDevice(h_c);

    float finalValue = 0.0;

    // verify that the GPU did the work we requested
    for (int i = 0; i < BLOCKS; i++) {
        finalValue += h_c.getElement(i);
    }

    printf("Dot result = %f \n", finalValue);

    printf("-: successful execution :-\n");

    free(h_c.elements);
}

int main(void) {
    test();
    return 0;
}