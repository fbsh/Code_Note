#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "common/CpuTimer.h"
#include "common/Error.h"
#include "common/GpuTimer.h"
#include "common/Matrix.h"

#define N 1024
#define K 16

void compareResults(Matrix<int> h_a, Matrix<int> h_b) {
    int i, j;

    for (i = 0; i < h_a.width; i++) {
        for (j = 0; j < h_a.height; j++) {
            assert(h_a.elements[j * h_a.width + i] ==
                   h_b.elements[i * h_b.width + j]);
        }
    }
}

__global__ void transposedMatrixKernelFinal(Matrix<int> d_a, Matrix<int> d_b) {
    // (i,j) locations of the tile corners for input & output matrices:
    int in_corner_i = blockIdx.x * blockDim.x,
        in_corner_j = blockIdx.y * blockDim.y;
    int out_corner_i = blockIdx.y * blockDim.y,
        out_corner_j = blockIdx.x * blockDim.x;

    int x = threadIdx.x, y = threadIdx.y;

    __shared__ float tile[K][K + 1];

    while (in_corner_j + x < N) {
        in_corner_i = blockIdx.x * blockDim.x;
        out_corner_j = blockIdx.x * blockDim.x;
        while (in_corner_i + y < N) {
            tile[y][x] =
                d_a.elements[(in_corner_i + x) + (in_corner_j + y) * N];

            __syncthreads();

            d_b.elements[(out_corner_i + x) + (out_corner_j + y) * N] =
                tile[x][y];

            in_corner_i += blockDim.x * gridDim.x;
            out_corner_j += blockDim.x * gridDim.x;
        }
        in_corner_j += gridDim.y * blockDim.y;
        out_corner_i += gridDim.y * blockDim.y;
    }
}

__global__ void transposedMatrixKernel_tile_padded(Matrix<int> d_a,
                                                   Matrix<int> d_b) {
    // (i,j) locations of the tile corners for input & output matrices:
    int in_corner_i = blockIdx.x * blockDim.x,
        in_corner_j = blockIdx.y * blockDim.y;
    int out_corner_i = blockIdx.y * blockDim.y,
        out_corner_j = blockIdx.x * blockDim.x;

    int x = threadIdx.x, y = threadIdx.y;

    __shared__ float tile[K][K + 1];

    // coalesced read from global mem, TRANSPOSED write into shared mem:
    tile[y][x] = d_a.elements[(in_corner_i + x) + (in_corner_j + y) * N];
    __syncthreads();
    // read from shared mem, coalesced write to global mem:
    d_b.elements[(out_corner_i + x) + (out_corner_j + y) * N] = tile[x][y];
}

// Kernel v4 using K threads and N/K blocks
__global__ void transposedMatrixKernel_tile(Matrix<int> d_a, Matrix<int> d_b) {
    // (i,j) locations of the tile corners for input & output matrices:
    int in_corner_i = blockIdx.x * blockDim.x,
        in_corner_j = blockIdx.y * blockDim.y;
    int out_corner_i = blockIdx.y * blockDim.y,
        out_corner_j = blockIdx.x * blockDim.x;

    int x = threadIdx.x, y = threadIdx.y;

    __shared__ float tile[K][K];
    // coalesced read from global mem, TRANSPOSED write into shared mem:
    tile[y][x] = d_a.elements[(in_corner_i + x) + (in_corner_j + y) * N];
    __syncthreads();
    // read from shared mem, coalesced write to global mem:
    d_b.elements[(out_corner_i + x) + (out_corner_j + y) * N] = tile[x][y];
}

// Kernel v3 using K threads and N/K blocks
// Try this example with 8, 16 and 32 threads by block
__global__ void transposedMatrixKernel_threads_blocks(Matrix<int> d_a,
                                                      Matrix<int> d_b) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    d_b.setElement(i, j, d_a.getElement(j, i));
}

// Kernel v2 using the max number of threads in 1 block
__global__ void transposedMatrixKernel_threads(Matrix<int> d_a,
                                               Matrix<int> d_b,
                                               int THREADS) {
    int i = threadIdx.x;
    int j = 0;

    while (i < N) {
        while (j < N) {
            d_b.setElement(i, j, d_a.getElement(j, i));
            j++;
        }
        i += THREADS;
    }
}

// Kernel v1 using 1 thread and 1 block
__global__ void transposedMatrixKernel(Matrix<int> d_a, Matrix<int> d_b) {
    int i = 0;
    int j = 0;

    while (i < d_a.width) {
        j = 0;
        while (j < d_a.height) {
            d_b.setElement(i, j, d_a.getElement(j, i));
            j++;
        }
        i++;
    }
}

// Host function
void transposedMatrixHost(Matrix<int> d_a, Matrix<int> d_b) {
    // start timer
    CpuTimer timer;
    timer.Start();

    int i, j;

    for (i = 0; i < d_a.width; i++) {
        for (j = 0; j < d_a.height; j++) {
            d_b.setElement(i, j, d_a.getElement(j, i));
        }
    }
    // stop timer
    timer.Stop();

    // print time
    printf("Time Host:  %f ms\n", timer.Elapsed());
}

void onDevice(Matrix<int> h_a, Matrix<int> h_b) {
    // declare GPU data
    Matrix<int> d_a, d_b;
    d_a.width = h_a.width;
    d_a.height = h_a.height;

    d_b.width = h_b.width;
    d_b.height = h_b.height;

    const int ARRAY_BYTES = d_a.width * d_a.height * sizeof(int);

    // allocate  memory on the GPU
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_b.elements, ARRAY_BYTES));

    // copy data from CPU the GPU
    HANDLER_ERROR_ERR(hipMemcpy(d_a.elements, h_a.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(hipMemcpy(d_b.elements, h_b.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));

    GpuTimer timer;

    // -*- [1] -*-
    timer.Start();
    transposedMatrixKernel<<<1, 1>>>(d_a, d_b);
    HANDLER_ERROR_MSG("kernel panic!!!");
    timer.Stop();
    printf("Time Device serial:  %f ms\n", timer.Elapsed());
    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_b.elements, d_b.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));
    compareResults(h_a, h_b);

    // -*- [2] -*-
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int THREADS = prop.maxThreadsPerBlock;
    timer.Start();
    transposedMatrixKernel_threads<<<1, THREADS>>>(d_a, d_b, THREADS);
    timer.Stop();
    printf("Time Device threads:  %f ms\n", timer.Elapsed());
    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_b.elements, d_b.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));
    compareResults(h_a, h_b);

    // -*- [3] -*-
    timer.Start();
    dim3 GridBlocks(N / K, N / K);
    dim3 ThreadsBlocks(K, K);
    transposedMatrixKernel_threads_blocks<<<GridBlocks, ThreadsBlocks>>>(d_a,
                                                                         d_b);
    HANDLER_ERROR_MSG("kernel panic!!!");
    timer.Stop();
    printf("Time Device threads and blocks:  %f ms\n", timer.Elapsed());
    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_b.elements, d_b.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));
    compareResults(h_a, h_b);

    // -*- [4] -*-
    timer.Start();
    transposedMatrixKernel_tile<<<GridBlocks, ThreadsBlocks>>>(d_a, d_b);
    HANDLER_ERROR_MSG("kernel panic!!!");
    timer.Stop();
    printf("Time Device tile:  %f ms\n", timer.Elapsed());
    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_b.elements, d_b.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));
    compareResults(h_a, h_b);

    // -*- [5] -*-
    timer.Start();
    transposedMatrixKernel_tile_padded<<<GridBlocks, ThreadsBlocks>>>(d_a, d_b);
    HANDLER_ERROR_MSG("kernel panic!!!");
    timer.Stop();
    printf("Time Device tile-padded:  %f ms\n", timer.Elapsed());
    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_b.elements, d_b.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));
    compareResults(h_a, h_b);
    // h_b.print();

    // dim3 GridBlocks1( 4,4 );
    // dim3 ThreadsBlocks1( 2,2 );

    // -*- [6] -*-
    timer.Start();
    transposedMatrixKernelFinal<<<GridBlocks, ThreadsBlocks>>>(d_a, d_b);
    HANDLER_ERROR_MSG("kernel panic!!!");
    timer.Stop();
    printf("Time Device final:  %f ms\n", timer.Elapsed());
    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_b.elements, d_b.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));
    compareResults(h_a, h_b);

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_a.elements));
    HANDLER_ERROR_ERR(hipFree(d_b.elements));
}

void test(Matrix<int> h_a, Matrix<int> h_b) {
    transposedMatrixHost(h_a, h_b);
    compareResults(h_a, h_b);
}

void onHost() {
    Matrix<int> h_a, h_b, h_c;
    h_a.width = N;
    h_a.height = N;

    h_b.width = N;
    h_b.height = N;

    h_c.width = N;
    h_c.height = N;

    h_a.elements = (int*)malloc(h_a.width * h_b.height * sizeof(int));
    h_b.elements = (int*)malloc(h_b.width * h_b.height * sizeof(int));
    h_c.elements = (int*)malloc(h_b.width * h_b.height * sizeof(int));

    int i, j, k = 0;

    for (i = 0; i < h_a.width; i++) {
        for (j = 0; j < h_a.height; j++) {
            h_a.elements[j * h_a.width + i] = k;
            h_b.elements[j * h_b.width + i] = k;
            k++;
        }
    }

    // call host function
    test(h_a, h_b);

    // call device configuration
    onDevice(h_a, h_c);

    printf("-: successful execution :-\n");
}

int main() {
    onHost();
}
