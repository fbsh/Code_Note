#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include "common/Error.h"
#include "common/GpuTimer.h"
#include "common/Vector.h"

#define N 500
#define FULL_DATA_SIZE 100000000
#define DIMGRID 10
#define DIMBLOCK 10
#define XMIN -10.0f
#define XMAX 10.0f

const int ARRAY_BYTES = N * sizeof(float);
const int FULL_ARRAY_BYTES = FULL_DATA_SIZE * sizeof(float);

__host__ __device__ float function1(float x) {
    return x * x;
}

__host__ __device__ float function2(float x) {
    return sinf(x);
}

__global__ void functionKernel1(Vector<float> d_a, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float x, dx;

    dx = (XMAX - (XMIN)) / ((float)N - 1);
    while (i < n) {
        x = XMIN + i * dx;
        d_a.setElement(i, function1(x));
        i += blockDim.x * gridDim.x;
    }
}

__global__ void functionKernel2(Vector<float> d_a, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float x, dx;

    dx = (XMAX - (XMIN)) / ((float)N - 1);

    while (i < n) {
        x = XMIN + i * dx;
        d_a.setElement(i, function2(x));
        i += blockDim.x * gridDim.x;
    }
}

void onDevice(Vector<float> h_a, Vector<float> h_b) {
    Vector<float> d_a, d_b;

    // create the stream
    hipStream_t stream1;
    HANDLER_ERROR_ERR(hipStreamCreate(&stream1));

    GpuTimer timer;
    timer.Start(stream1);

    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_b.elements, ARRAY_BYTES));

    for (int i = 0; i < FULL_DATA_SIZE; i += N) {
        // copy the locked memory to the device, async
        HANDLER_ERROR_ERR(hipMemcpyAsync(d_a.elements, h_a.elements + i,
                                          ARRAY_BYTES, hipMemcpyHostToDevice,
                                          stream1));

        // copy the locked memory to the device, async
        HANDLER_ERROR_ERR(hipMemcpyAsync(d_b.elements, h_b.elements + i,
                                          ARRAY_BYTES, hipMemcpyHostToDevice,
                                          stream1));

        functionKernel1<<<DIMGRID, DIMBLOCK, 0, stream1>>>(d_a, N);
        HANDLER_ERROR_MSG("kernel panic!!!");

        functionKernel2<<<DIMGRID, DIMBLOCK, 0, stream1>>>(d_b, N);
        HANDLER_ERROR_MSG("kernel panic!!!");

        /*
        HANDLER_ERROR_ERR(hipMemcpy(h_a.elements, d_a.elements, ARRAY_BYTES,
        hipMemcpyDeviceToHost)); HANDLER_ERROR_ERR(hipMemcpy(h_b.elements,
        d_b.elements, ARRAY_BYTES, hipMemcpyDeviceToHost));
        */

        // copy from the device to the locked memory, async
        HANDLER_ERROR_ERR(hipMemcpyAsync(h_a.elements + i, d_a.elements,
                                          ARRAY_BYTES, hipMemcpyDeviceToHost,
                                          stream1));

        HANDLER_ERROR_ERR(hipMemcpyAsync(h_b.elements + i, d_b.elements,
                                          ARRAY_BYTES, hipMemcpyDeviceToHost,
                                          stream1));
    }

    // synchronization
    HANDLER_ERROR_ERR(hipStreamSynchronize(stream1));

    // stop timer
    timer.Stop(stream1);

    // print time
    printf("Time :  %f ms\n", timer.Elapsed());

    // destroy stream
    HANDLER_ERROR_ERR(hipStreamDestroy(stream1));

    // free device memory
    HANDLER_ERROR_ERR(hipFree(d_a.elements));
    HANDLER_ERROR_ERR(hipFree(d_b.elements));
}

void checkDeviceProps() {
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLER_ERROR_ERR(hipGetDevice(&whichDevice));
    HANDLER_ERROR_ERR(hipGetDeviceProperties(&prop, whichDevice));
    if (!prop.deviceOverlap) {
        printf(
            "Device will not handle overlaps, so no speed up from streams\n");
    }
}

void test() {
    Vector<float> h_a, h_b;
    h_a.length = FULL_DATA_SIZE;
    h_b.length = FULL_DATA_SIZE;

    /* Not used because of the hipHostAlloc
    h_a.elements = (float*)malloc( ARRAY_BYTES );
    h_b.elements = (float*)malloc( ARRAY_BYTES );
    */

    // allocate host locked memory
    HANDLER_ERROR_ERR(hipHostAlloc((void**)&h_a.elements, FULL_ARRAY_BYTES,
                                    hipHostMallocDefault));
    HANDLER_ERROR_ERR(hipHostAlloc((void**)&h_b.elements, FULL_ARRAY_BYTES,
                                    hipHostMallocDefault));

    int i;
    for (i = 0; i < FULL_DATA_SIZE; i++) {
        h_a.setElement(i, 1.0);
        h_b.setElement(i, 1.0);
    }

    // call device configuration
    onDevice(h_a, h_b);

    /* Not used because of the hipHostAlloc
    free(h_a.elements);
    free(h_b.elements);
    */
    for (i = 0; i < FULL_DATA_SIZE; i++) {
        assert(h_a.getElement(i) != 1);
        assert(h_b.getElement(i) != 1);
    }

    printf("-: successful execution :-\n");

    // free host memory
    HANDLER_ERROR_ERR(hipHostFree(h_a.elements));
    HANDLER_ERROR_ERR(hipHostFree(h_b.elements));
}

int main() {
    checkDeviceProps();
    test();
}
