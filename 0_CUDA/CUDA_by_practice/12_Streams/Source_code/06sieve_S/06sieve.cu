#include "hip/hip_runtime.h"
#include "common/Error.h"
#include "common/GpuTimer.h"
#include "common/Vector.h"

#define MARK 1
#define UNMARK 0
#define ARRAY_SIZE 16384

const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

__global__ void kernelSieve(int k, Vector<int> d_a) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    while (i < ARRAY_SIZE) {
        if (k * k <= i) {
            if (i % k == 0)
                d_a.setElement(i, MARK);
        }
        i += blockDim.x * gridDim.x;
    }
}

void onDevice(Vector<int> h_a) {
    Vector<int> d_a;
    int k;
    // create the stream
    hipStream_t stream1;
    HANDLER_ERROR_ERR(hipStreamCreate(&stream1));

    HANDLER_ERROR_ERR(hipMalloc(&d_a.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMemcpy(d_a.elements, h_a.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));

    for (k = 2; k <= ARRAY_SIZE; k++) {
        kernelSieve<<<64, 256, 0, stream1>>>(k, d_a);
    }

    HANDLER_ERROR_ERR(hipMemcpy(h_a.elements, d_a.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));
    HANDLER_ERROR_ERR(hipFree(d_a.elements));

    // destroy stream
    HANDLER_ERROR_ERR(hipStreamDestroy(stream1));
}

void onHost() {
    Vector<int> h_a;
    h_a.length = ARRAY_SIZE;

    int j;
    h_a.elements = (int*)malloc(ARRAY_BYTES);

    for (j = 0; j < ARRAY_SIZE; j++) {
        h_a.setElement(j, j);
    }

    onDevice(h_a);

    for (j = 0; j < ARRAY_SIZE; j++) {
        if (h_a.getElement(j) > 1)
            printf("%i \n", h_a.getElement(j));
    }

    free(h_a.elements);
}

void checkDeviceProps() {
    // properties validation
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLER_ERROR_ERR(hipGetDevice(&whichDevice));
    HANDLER_ERROR_ERR(hipGetDeviceProperties(&prop, whichDevice));
    if (!prop.deviceOverlap) {
        printf(
            "Device will not handle overlaps, so no speed up from streams\n");
    }
}

int main() {
    checkDeviceProps();
    onHost();
}
