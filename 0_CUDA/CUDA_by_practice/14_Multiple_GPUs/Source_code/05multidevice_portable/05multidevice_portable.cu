#include "hip/hip_runtime.h"
#include <hiprand.h>
#include "common/CudaPthread.h"
#include "common/Error.h"
#include "common/GpuTimer.h"
#include "common/Vector.h"

#define N (32 * 1024)  // (32768/2)=16384
#define THREADS 256
#define BLOCKS 32

const int ARRAY_BYTES = N * sizeof(float);
const int P_ARRAY_BYTES = BLOCKS * sizeof(float);

struct DataStruct {
    int deviceID;
    int size;
    int offset;
    float* a;
    float* b;
    float returnValue;
};

__global__ void dotKernel(Vector<float> d_a,
                          Vector<float> d_b,
                          Vector<float> d_c,
                          int size) {
    __shared__ float cache[THREADS];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < size) {
        temp += d_a.getElement(tid) * d_b.getElement(tid);
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cacheIndex] = temp;

    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x / 2;
    while (i != 0) {
        __syncthreads();
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        i /= 2;
    }

    if (cacheIndex == 0)
        d_c.setElement(blockIdx.x, cache[0]);
}

void* onDevice(void* pvoidData) {
    DataStruct* data = (DataStruct*)pvoidData;
    if (data->deviceID != 0) {
        HANDLER_ERROR_ERR(hipSetDevice(data->deviceID));
        HANDLER_ERROR_ERR(hipSetDeviceFlags(hipDeviceMapHost));
    }

    const int PARTIAL_ARRAY_SIZE = data->size;

    Vector<float> h_a, h_b, h_c;
    Vector<float> d_a, d_b, d_c;
    d_c.length = BLOCKS;

    // allocate memory on the CPU side
    h_a.elements = data->a;
    h_b.elements = data->b;
    h_c.elements = (float*)malloc(P_ARRAY_BYTES);

    // start timer
    GpuTimer timer;
    timer.Start();

    // allocate  memory on the GPU
    HANDLER_ERROR_ERR(hipHostGetDevicePointer(&d_a.elements, h_a.elements, 0));
    HANDLER_ERROR_ERR(hipHostGetDevicePointer(&d_b.elements, h_b.elements, 0));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_c.elements, P_ARRAY_BYTES));

    // offset 'a' and 'b' to where this GPU is getting its data
    d_a.elements += data->offset;
    d_b.elements += data->offset;

    dotKernel<<<BLOCKS, THREADS>>>(d_a, d_b, d_c, PARTIAL_ARRAY_SIZE);
    HANDLER_ERROR_MSG("kernel panic!!!");

    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_c.elements, d_c.elements, P_ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));

    // finish up on the CPU side
    float partial = 0;
    for (int i = 0; i < BLOCKS; i++) {
        partial += h_c.getElement(i);
    }

    // stop timer
    timer.Stop();

    // print time
    printf("Time :  %f ms\n", timer.Elapsed());

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_c.elements));

    // free CPU memory
    free(h_c.elements);
    data->returnValue = partial;
    return 0;
}

void test() {
    Vector<float> h_a, h_b;
    h_a.length = N;
    h_b.length = N;

    HANDLER_ERROR_ERR(hipSetDevice(0));
    HANDLER_ERROR_ERR(hipSetDeviceFlags(hipDeviceMapHost));
    HANDLER_ERROR_ERR(hipHostAlloc((void**)&h_a.elements, ARRAY_BYTES,
                                    hipHostMallocWriteCombined |
                                        hipHostMallocPortable |
                                        hipHostMallocMapped));

    HANDLER_ERROR_ERR(hipHostAlloc((void**)&h_b.elements, ARRAY_BYTES,
                                    hipHostMallocWriteCombined |
                                        hipHostMallocPortable |
                                        hipHostMallocMapped));

    int i;
    for (i = 0; i < N; i++) {
        h_a.setElement(i, 1.0);
        h_b.setElement(i, 1.0);
    }

    // prepare for multithread
    DataStruct data[2];
    data[0].deviceID = 0;
    data[0].offset = 0;
    data[0].size = N / 2;
    data[0].a = h_a.elements;
    data[0].b = h_b.elements;

    data[1].deviceID = 1;
    data[1].offset = N / 2;
    data[1].size = N / 2;
    data[1].a = h_a.elements;
    data[1].b = h_b.elements;

    CudaPthread thread;
    thread.start_thread(onDevice, &(data[1]));
    onDevice(&(data[0]));
    thread.end_thread();

    float finalValue = data[0].returnValue + data[1].returnValue;

    printf("Dot result = %f \n", finalValue);

    printf("-: successful execution :-\n");

    HANDLER_ERROR_ERR(hipHostFree(h_a.elements));
    HANDLER_ERROR_ERR(hipHostFree(h_b.elements));
}

void checkDeviceProps() {
    // properties validation
    int deviceCount;
    HANDLER_ERROR_ERR(hipGetDeviceCount(&deviceCount));
    if (deviceCount < 2) {
        printf(
            "We need at least two compute 1.0 or greater "
            "devices, but only found %d\n",
            deviceCount);
    }

    hipDeviceProp_t prop;
    for (int i = 0; i < deviceCount; i++) {
        HANDLER_ERROR_ERR(hipGetDeviceProperties(&prop, i));
        if (prop.canMapHostMemory != 1) {
            printf("Device %d can not map memory.\n", i);
        }
    }
}

int main(void) {
    checkDeviceProps();
    test();
    return 0;
}
