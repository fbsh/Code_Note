
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("Hello world from device\n");
}

int main() {
    hello<<<1, 1>>>();
    printf("Hello world from host\n");
    hipDeviceSynchronize();
    return 0;
}
