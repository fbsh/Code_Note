#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "Error.h"

__global__ void cubeKernel(float* d_out, float* d_in) {
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}

void onDevice(float* h_in, float* h_out, int ARRAY_SIZE, int ARRAY_BYTES) {
    // declare GPU memory pointers
    float* d_in;
    float* d_out;

    // allocate GPU memory
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_in, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_out, ARRAY_BYTES));

    // transfer the array to the GPU
    HANDLER_ERROR_ERR(
        hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice));

    // launch the kernel
    cubeKernel<<<1, ARRAY_SIZE>>>(d_out, d_in);
    HANDLER_ERROR_MSG("Kernel Panic!!!");

    // copy back the result array to the CPU
    HANDLER_ERROR_ERR(
        hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));

    // free GPU memory pointers
    HANDLER_ERROR_ERR(hipFree(d_in));
    HANDLER_ERROR_ERR(hipFree(d_out));
}

void test(float* h_in, float* h_out, int ARRAY_SIZE, int ARRAY_BYTES) {
    // print out the resulting array
    for (int i = 0; i < ARRAY_SIZE; i++) {
        assert(h_out[i] == (h_in[i] * h_in[i] * h_in[i]));
        printf("%f", h_out[i]);
        printf(((i % 4) != 3) ? "\t" : "\n");
    }

    printf("-: successful execution :-\n");
}

void onHost() {
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // declare CPU memory pointers
    float* h_in;
    float* h_out;

    // allocate CPU memory
    h_in = (float*)malloc(ARRAY_BYTES);
    h_out = (float*)malloc(ARRAY_BYTES);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }

    // call the kernel
    onDevice(h_in, h_out, ARRAY_SIZE, ARRAY_BYTES);
    test(h_in, h_out, ARRAY_SIZE, ARRAY_BYTES);

    // free CPU memory pointers
    free(h_in);
    free(h_out);
}

int main(int argc, char** argv) {
    onHost();

    return 0;
}