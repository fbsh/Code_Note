#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "Error.h"

#define N 1000

__global__ void transposedMatrixKernel(int* d_a, int* d_b) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    while (i < N) {
        j = threadIdx.y + blockDim.y * blockIdx.y;
        while (j < N) {
            d_b[i * N + j] = d_a[j * N + i];
            j += blockDim.y * gridDim.y;
        }
        i += blockDim.x * gridDim.x;
    }
}

void onDevice(int h_a[][N], int h_b[][N]) {
    // declare GPU memory pointers
    int *d_a, *d_b;

    const int ARRAY_BYTES = N * N * sizeof(int);

    // allocate  memory on the GPU
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_b, ARRAY_BYTES));

    // copy data from CPU the GPU
    HANDLER_ERROR_ERR(
        hipMemcpy(d_a, h_a, ARRAY_BYTES, hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(
        hipMemcpy(d_b, h_b, ARRAY_BYTES, hipMemcpyHostToDevice));

    // execution configuration
    dim3 GridBlocks(4, 4);
    dim3 ThreadsBlocks(16, 16);

    // run the kernel
    transposedMatrixKernel<<<GridBlocks, ThreadsBlocks>>>(d_a, d_b);
    HANDLER_ERROR_MSG("kernel panic!!!");

    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(
        hipMemcpy(h_b, d_b, ARRAY_BYTES, hipMemcpyDeviceToHost));

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_a));
    HANDLER_ERROR_ERR(hipFree(d_b));
}

void test(int h_a[][N], int h_b[][N]) {
    // test  result
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            assert(h_a[j][i] == h_b[i][j]);
        }
    }

    printf("-: successful execution :-\n");
}

void onHost() {
    int i, j, k = 0;
    int h_a[N][N], h_b[N][N];

    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            h_a[i][j] = k;
            h_b[i][j] = 0;
            k++;
        }
    }

    // call device configuration
    onDevice(h_a, h_b);
    test(h_a, h_b);
}

int main() {
    onHost();
}
