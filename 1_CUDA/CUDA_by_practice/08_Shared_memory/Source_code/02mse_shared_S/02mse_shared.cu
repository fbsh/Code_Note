#include "hip/hip_runtime.h"
#include <assert.h>
#include "Error.h"
#include "GpuTimer.h"
#include "Vector.h"

#define N 16
#define BLOCK_SIZE 2
#define STRIDE 4
#define POW(x) (x) * (x)

__global__ void mseKernel(Vector<float> d_a,
                          Vector<float> d_b,
                          Vector<float> d_c) {
    __shared__ float cache[4];

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index (current coefficient)
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Indixes
    int idx = (by * BLOCK_SIZE + ty) * STRIDE + (bx * BLOCK_SIZE + tx);
    int ith = ty * BLOCK_SIZE + tx;

    // operation
    cache[ith] = POW(d_a.getElement(idx) - d_b.getElement(idx));

    __syncthreads();

    int i = 2;
    while (i != 0) {
        if (ith < i)
            cache[ith] += cache[ith + i];
        __syncthreads();
        i /= 2;
    }

    int bidx = by * BLOCK_SIZE + bx;
    if (ith == 0)
        d_c.setElement(bidx, cache[0]);
}

void onDevice(Vector<float> h_a, Vector<float> h_b, Vector<float> h_c) {
    Vector<float> d_a, d_b, d_c;
    d_a.length = N;
    d_b.length = N;
    d_c.length = 4;

    // start timer
    GpuTimer timer;
    timer.Start();

    const int ARRAY_BYTES = N * sizeof(float);

    // allocate  memory on the GPU
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_b.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_c.elements, 4 * sizeof(float)));

    // copy data from CPU the GPU
    HANDLER_ERROR_ERR(hipMemcpy(d_a.elements, h_a.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(hipMemcpy(d_b.elements, h_b.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));

    // execution configuration
    dim3 GridBlocks(2, 2);
    dim3 ThreadsBlocks(2, 2);

    mseKernel<<<GridBlocks, ThreadsBlocks>>>(d_a, d_b, d_c);
    HANDLER_ERROR_MSG("kernel panic!!!");

    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_c.elements, d_c.elements, 4 * sizeof(float),
                                 hipMemcpyDeviceToHost));

    // stop timer
    timer.Stop();

    // print time
    printf("Time :  %f ms\n", timer.Elapsed());

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_a.elements));
    HANDLER_ERROR_ERR(hipFree(d_b.elements));
    HANDLER_ERROR_ERR(hipFree(d_c.elements));
}

void test() {
    Vector<float> h_a, h_b, h_c;
    h_a.length = N;
    h_b.length = N;
    h_c.length = 4;

    h_a.elements = (float*)malloc(h_a.length * sizeof(float));
    h_b.elements = (float*)malloc(h_a.length * sizeof(float));
    h_c.elements = (float*)malloc(4 * sizeof(float));

    int i, j = 16, k = 1;

    for (i = 0; i < h_a.length; i++) {
        h_a.setElement(i, k);
        h_b.setElement(i, j);
        k++;
        j--;
    }

    // call device configuration
    onDevice(h_a, h_b, h_c);

    // verify that the GPU did the work we requested
    float d_mse = 0, h_mse = 0;
    for (int i = 0; i < 4; i++) {
        d_mse += h_c.getElement(i);
        printf(" [%i] = %f \n", i, h_c.getElement(i));
    }
    d_mse /= N;
    printf("mse from device: %f\n", d_mse);

    for (int i = 0; i < N; i++) {
        h_mse += POW(h_a.getElement(i) - h_b.getElement(i));
    }
    h_mse /= N;
    printf("mse from host: %f\n", h_mse);

    assert(d_mse == h_mse);

    printf("-: successful execution :-\n");

    free(h_a.elements);
    free(h_b.elements);
    free(h_c.elements);
}

int main(void) {
    test();
    return 0;
}
