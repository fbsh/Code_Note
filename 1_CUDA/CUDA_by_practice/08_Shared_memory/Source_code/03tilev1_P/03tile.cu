#include "hip/hip_runtime.h"
#include <assert.h>
#include "Error.h"
#include "GpuTimer.h"
#include "Vector.h"

const int BLOCKSIZE = 128;
const int NUMBLOCKS = 1000;
const int N = BLOCKSIZE * NUMBLOCKS;
const int ARRAY_BYTES = N * sizeof(int);

__global__ void tileKernelv1(Vector<int> d_a,
                             Vector<int> d_b,
                             Vector<int> d_c,
                             Vector<int> d_d,
                             Vector<int> d_e,
                             Vector<int> d_out) {
    // Change next operation in order to use the tiling technique
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    d_out.elements[i] =
        (d_a.getElement(i) + d_b.getElement(i) + d_c.getElement(i) +
         d_d.getElement(i) + d_e.getElement(i)) /
        5.0f;

    // -:YOUR CODE HERE:-
}

void onDevice(Vector<int> h_a,
              Vector<int> h_b,
              Vector<int> h_c,
              Vector<int> h_d,
              Vector<int> h_e,
              Vector<int> h_out) {
    Vector<int> d_a, d_b, d_c, d_d, d_e, d_out;
    d_a.length = N;
    d_b.length = N;
    d_c.length = N;
    d_d.length = N;
    d_e.length = N;
    d_out.length = N;

    // allocate  memory on the GPU
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_b.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_c.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_d.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_e.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_out.elements, ARRAY_BYTES));

    // copy data from CPU the GPU
    HANDLER_ERROR_ERR(hipMemcpy(d_a.elements, h_a.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(hipMemcpy(d_b.elements, h_b.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(hipMemcpy(d_c.elements, h_b.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(hipMemcpy(d_d.elements, h_b.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(hipMemcpy(d_e.elements, h_b.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));

    // launch kernel
    tileKernelv1<<<N / BLOCKSIZE, BLOCKSIZE>>>(d_a, d_b, d_c, d_d, d_e, d_out);
    HANDLER_ERROR_MSG("kernel panic!!!");

    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_out.elements, d_out.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_a.elements));
    HANDLER_ERROR_ERR(hipFree(d_b.elements));
    HANDLER_ERROR_ERR(hipFree(d_c.elements));
    HANDLER_ERROR_ERR(hipFree(d_d.elements));
    HANDLER_ERROR_ERR(hipFree(d_e.elements));
    HANDLER_ERROR_ERR(hipFree(d_out.elements));
}

void test(Vector<int> h_a,
          Vector<int> h_b,
          Vector<int> h_c,
          Vector<int> h_d,
          Vector<int> h_e,
          Vector<int> h_out) {
    int aux = 0;
    for (int i = 0; i < N - 2; i++) {
        aux = (h_a.getElement(i) + h_b.getElement(i) + h_c.getElement(i) +
               h_d.getElement(i) + h_e.getElement(i)) /
              5.0f;
        assert(aux == h_out.getElement(i + 2));
    }
}

void onHost() {
    Vector<int> h_a, h_b, h_c, h_d, h_e, h_out;
    h_a.length = N;
    h_b.length = N;
    h_c.length = N;
    h_d.length = N;
    h_e.length = N;
    h_out.length = N;

    h_a.elements = (int*)malloc(ARRAY_BYTES);
    h_b.elements = (int*)malloc(ARRAY_BYTES);
    h_c.elements = (int*)malloc(ARRAY_BYTES);
    h_d.elements = (int*)malloc(ARRAY_BYTES);
    h_e.elements = (int*)malloc(ARRAY_BYTES);
    h_out.elements = (int*)malloc(ARRAY_BYTES);

    for (int i = 0; i < N; i++) {
        h_a.setElement(i, i);
        h_b.setElement(i, i + 1);
        h_c.setElement(i, i + 2);
        h_d.setElement(i, i + 3);
        h_e.setElement(i, i + 4);
    }

    // call device configuration
    onDevice(h_a, h_b, h_c, h_d, h_e, h_out);

    test(h_a, h_b, h_c, h_d, h_e, h_out);

    printf("-: successful execution :-\n");

    free(h_a.elements);
    free(h_b.elements);
    free(h_c.elements);
    free(h_d.elements);
    free(h_e.elements);
    free(h_out.elements);
}

int main(void) {
    onHost();
    return 0;
}