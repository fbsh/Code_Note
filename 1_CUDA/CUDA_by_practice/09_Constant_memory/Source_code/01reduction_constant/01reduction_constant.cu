#include "hip/hip_runtime.h"
#include <assert.h>
#include "Error.h"
#include "GpuTimer.h"
#include "Vector.h"

// check the constant memory amount

#define N (16 * 1024)
#define THREADS 256
#define BLOCKS 32

const int ARRAY_BYTES = N * sizeof(float);
const int P_ARRAY_BYTES = BLOCKS * sizeof(float);

// constant Vectors in GPU
__constant__ float d_a[N];
__constant__ float d_b[N];

__global__ void dotKernel(Vector<float> d_c) {
    __shared__ float cache[THREADS];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < N) {
        temp += d_a[tid] * d_b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cacheIndex] = temp;

    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x / 2;
    while (i != 0) {
        __syncthreads();
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        i /= 2;
    }

    if (cacheIndex == 0)
        d_c.setElement(blockIdx.x, cache[0]);
}

void onDevice(Vector<float> h_a, Vector<float> h_b, Vector<float> h_c) {
    Vector<float> d_c;
    d_c.length = BLOCKS;

    // start timer
    GpuTimer timer;
    timer.Start();

    int i;
    for (i = 0; i < 2; i++) {
        // allocate  memory on the GPU
        HANDLER_ERROR_ERR(hipMemcpyToSymbol(HIP_SYMBOL(d_a), h_a.elements, ARRAY_BYTES));
        HANDLER_ERROR_ERR(hipMemcpyToSymbol(HIP_SYMBOL(d_b), h_b.elements, ARRAY_BYTES));
        HANDLER_ERROR_ERR(hipMalloc((void**)&d_c.elements, P_ARRAY_BYTES));

        dotKernel<<<BLOCKS, THREADS>>>(d_c);
        HANDLER_ERROR_MSG("kernel panic!!!");

        // copy data back from the GPU to the CPU
        HANDLER_ERROR_ERR(hipMemcpy(h_c.elements, d_c.elements, P_ARRAY_BYTES,
                                     hipMemcpyDeviceToHost));
    }

    // stop timer
    timer.Stop();

    // print time
    printf("Time :  %f ms\n", timer.Elapsed());

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_c.elements));
}

void test() {
    Vector<float> h_a, h_b, h_c;
    h_a.length = N;
    h_b.length = N;
    h_c.length = BLOCKS;

    h_a.elements = (float*)malloc(ARRAY_BYTES);
    h_b.elements = (float*)malloc(ARRAY_BYTES);
    h_c.elements = (float*)malloc(P_ARRAY_BYTES);

    int i;

    for (i = 0; i < h_a.length; i++) {
        h_a.setElement(i, 1.0);
        h_b.setElement(i, 1.0);
    }

    // call device configuration
    onDevice(h_a, h_b, h_c);
    float d_dot_result = 0.0;

    // verify that the GPU did the work we requested
    for (int i = 0; i < BLOCKS; i++) {
        d_dot_result += h_c.getElement(i);
    }

    printf("Dot result from device = %f \n", d_dot_result);

    float h_dot_result = 0.0;
    for (int i = 0; i < N; i++) {
        h_dot_result += h_a.getElement(i) * h_b.getElement(i);
    }

    printf("Dot result from host = %f \n", h_dot_result);

    assert(d_dot_result == h_dot_result);

    printf("-: successful execution :-\n");

    free(h_a.elements);
    free(h_b.elements);
    free(h_c.elements);
}

int main(void) {
    test();
    return 0;
}
