#include "hip/hip_runtime.h"
#include <assert.h>
#include <time.h>
#include "common/CpuTimer.h"
#include "common/Error.h"
#include "common/GpuTimer.h"
#include "common/Vector.h"

#define SIZE (100 * 1024 * 1024)
#define SIZE_H 256

const int ARRAY_BYTES = SIZE * sizeof(unsigned int);
const int ARRAY_BYTES_H = SIZE_H * sizeof(int);

__global__ void histo_kernel(Vector<unsigned int> d_a, Vector<int> d_histo) {
    // clear out the accumulation buffer called cache
    // since we are launched with 256 threads, it is easy
    // to clear that memory with one write per thread
    __shared__ int cache[256];
    cache[threadIdx.x] = 0;
    __syncthreads();

    // calculate the starting index and the offset to the next
    // block that each thread will be processing
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int tmp = 0;
    while (i < SIZE) {
        tmp = d_a.getElement(i);
        atomicAdd(&cache[tmp], 1);
        i += stride;
    }
    // sync the data from the above writes to shared memory
    // then add the shared memory values to the values from
    // the other thread blocks using global memory
    // atomic adds
    // same as before, since we have 256 threads, updating the
    // global histogram is just one write per thread!
    __syncthreads();
    atomicAdd(&(d_histo.elements[threadIdx.x]), cache[threadIdx.x]);
}

void onDevice(Vector<unsigned int> h_a, Vector<int> h_histo) {
    Vector<unsigned int> d_a;
    d_a.length = SIZE;

    Vector<int> d_histo;
    d_histo.length = SIZE_H;

    // start timer
    GpuTimer timer;
    timer.Start();

    // allocate memory on the GPU for the file's data
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_histo.elements, ARRAY_BYTES_H));

    // copy data from CPU the GPU
    HANDLER_ERROR_ERR(hipMemcpy(d_a.elements, h_a.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(hipMemcpy(d_histo.elements, h_histo.elements,
                                 ARRAY_BYTES_H, hipMemcpyHostToDevice));

    // kernel launch - 2x the number of mps gave best timing
    hipDeviceProp_t prop;
    HANDLER_ERROR_ERR(hipGetDeviceProperties(&prop, 0));
    int blocks = prop.multiProcessorCount;

    histo_kernel<<<blocks * 2, SIZE_H>>>(d_a, d_histo);
    HANDLER_ERROR_MSG("kernel panic!!!");

    HANDLER_ERROR_ERR(hipMemcpy(h_histo.elements, d_histo.elements,
                                 ARRAY_BYTES_H, hipMemcpyDeviceToHost));

    // stop timer
    timer.Stop();

    // print time
    printf("GPU Time :  %f ms\n", timer.Elapsed());

    // free device memory
    hipFree(d_histo.elements);
    hipFree(d_a.elements);
}

void test(Vector<unsigned int> h_a, Vector<int> h_histo, Vector<int> h_test) {
    // start timer
    CpuTimer timer;
    timer.Start();

    for (int i = 0; i < SIZE; i++) {
        h_test.elements[h_a.elements[i]]++;
    }

    // stop timer
    timer.Stop();

    // print time
    printf("CPU Time :  %f ms\n", timer.Elapsed());

    for (int i = 0; i < SIZE_H; i++) {
        // printf(" [%i]  %i | %i \n", i, h_histo.getElement(i),
        // h_test.getElement(i));
        assert(h_histo.getElement(i) == h_test.getElement(i));
    }

    printf("-: successful execution :-\n");
}

void onHost() {
    Vector<unsigned int> h_a;
    h_a.length = SIZE;
    h_a.elements = (unsigned int*)malloc(ARRAY_BYTES);
    h_a.randomInit(0, 255);

    Vector<int> h_histo;
    h_histo.length = SIZE_H;
    h_histo.elements = (int*)malloc(ARRAY_BYTES_H);
    h_histo.zerosInit();

    Vector<int> h_test;
    h_test.length = SIZE_H;
    h_test.elements = (int*)malloc(ARRAY_BYTES_H);
    h_test.zerosInit();

    onDevice(h_a, h_histo);

    test(h_a, h_histo, h_test);

    free(h_a.elements);
    free(h_histo.elements);
    free(h_test.elements);
}

int main(void) {
    onHost();

    return 0;
}
