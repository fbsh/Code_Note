#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include "common/CpuTimer.h"
#include "common/Error.h"
#include "common/GpuTimer.h"
#include "common/Matrix.h"

#define N 2048
#define K 16

void compareResults(Matrix<int> h_a, Matrix<int> h_b) {
    int i, j;

    for (i = 0; i < h_a.width; i++) {
        for (j = 0; j < h_a.height; j++) {
            assert(h_a.elements[j * h_a.width + i] ==
                   h_b.elements[i * h_b.width + j]);
        }
    }
}

__global__ void transposedMatrixKernelFinal(Matrix<int> d_a, Matrix<int> d_b) {
    // (i,j) locations of the tile corners for input & output matrices:
    int in_corner_i = blockIdx.x * blockDim.x,
        in_corner_j = blockIdx.y * blockDim.y;
    int out_corner_i = blockIdx.y * blockDim.y,
        out_corner_j = blockIdx.x * blockDim.x;

    int x = threadIdx.x, y = threadIdx.y;

    __shared__ float tile[K][K];

    while (in_corner_j + x < N) {
        in_corner_i = blockIdx.x * blockDim.x;
        out_corner_j = blockIdx.x * blockDim.x;
        while (in_corner_i + y < N) {
            tile[y][x] =
                d_a.elements[(in_corner_i + x) + (in_corner_j + y) * N];

            __syncthreads();

            d_b.elements[(out_corner_i + x) + (out_corner_j + y) * N] =
                tile[x][y];

            in_corner_i += blockDim.x * gridDim.x;
            out_corner_j += blockDim.x * gridDim.x;
        }
        in_corner_j += gridDim.y * blockDim.y;
        out_corner_i += gridDim.y * blockDim.y;
    }
}

void onDevice(Matrix<int> h_a, Matrix<int> h_b) {
    // declare GPU data
    Matrix<int> d_a, d_b;
    d_a.width = h_a.width;
    d_a.height = h_a.height;

    d_b.width = h_b.width;
    d_b.height = h_b.height;

    const int ARRAY_BYTES = d_a.width * d_a.height * sizeof(int);

    // allocate  memory on the GPU
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_b.elements, ARRAY_BYTES));

    // copy data from CPU the GPU
    HANDLER_ERROR_ERR(hipMemcpy(d_a.elements, h_a.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));
    HANDLER_ERROR_ERR(hipMemcpy(d_b.elements, h_b.elements, ARRAY_BYTES,
                                 hipMemcpyHostToDevice));

    GpuTimer timer;

    dim3 GridBlocks(N / K, N / K);
    dim3 ThreadsBlocks(K, K);

    timer.Start();
    transposedMatrixKernelFinal<<<GridBlocks, ThreadsBlocks>>>(d_a, d_b);
    HANDLER_ERROR_MSG("kernel panic!!!");
    timer.Stop();
    printf("Time Device final:  %f ms\n", timer.Elapsed());
    // copy data back from the GPU to the CPU
    HANDLER_ERROR_ERR(hipMemcpy(h_b.elements, d_b.elements, ARRAY_BYTES,
                                 hipMemcpyDeviceToHost));
    compareResults(h_a, h_b);

    // free GPU memory
    HANDLER_ERROR_ERR(hipFree(d_a.elements));
    HANDLER_ERROR_ERR(hipFree(d_b.elements));
}

void onHost() {
    Matrix<int> h_a, h_b, h_c;
    h_a.width = N;
    h_a.height = N;

    h_b.width = N;
    h_b.height = N;

    h_c.width = N;
    h_c.height = N;

    h_a.elements = (int*)malloc(h_a.width * h_b.height * sizeof(int));
    h_b.elements = (int*)malloc(h_b.width * h_b.height * sizeof(int));
    h_c.elements = (int*)malloc(h_b.width * h_b.height * sizeof(int));

    int i, j, k = 0;

    for (i = 0; i < h_a.width; i++) {
        for (j = 0; j < h_a.height; j++) {
            h_a.elements[j * h_a.width + i] = k;
            h_b.elements[j * h_b.width + i] = 0;
            k++;
        }
    }

    // call device configuration
    onDevice(h_a, h_c);

    printf("-: successful execution :-\n");
}

int main() {
    onHost();
}
