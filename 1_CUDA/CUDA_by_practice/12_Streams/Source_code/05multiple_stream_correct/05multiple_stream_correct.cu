#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include "common/Error.h"
#include "common/GpuTimer.h"
#include "common/Vector.h"

#define N 500
#define FULL_DATA_SIZE 100000000
#define DIMGRID 10
#define DIMBLOCK 10
#define XMIN -10.0
#define XMAX 10.0

const int ARRAY_BYTES = N * sizeof(float);
const int FULL_ARRAY_BYTES = FULL_DATA_SIZE * sizeof(float);

__host__ __device__ float function1(float x) {
    return x * x;
}

__host__ __device__ float function2(float x) {
    return sinf(x);
}

__global__ void functionKernel1(Vector<float> d_a, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float x, dx;

    dx = (XMAX - (XMIN)) / ((float)N - 1);
    while (i < n) {
        x = XMIN + i * dx;
        d_a.setElement(i, function1(x));
        i += blockDim.x * gridDim.x;
    }
}

__global__ void functionKernel2(Vector<float> d_a, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float x, dx;

    dx = (XMAX - (XMIN)) / ((float)N - 1);

    while (i < n) {
        x = XMIN + i * dx;
        d_a.setElement(i, function2(x));
        i += blockDim.x * gridDim.x;
    }
}

void onDevice(Vector<float> h_a, Vector<float> h_b) {
    Vector<float> d_a, d_b;

    // create the stream
    hipStream_t stream1;
    hipStream_t stream2;
    HANDLER_ERROR_ERR(hipStreamCreate(&stream1));
    HANDLER_ERROR_ERR(hipStreamCreate(&stream2));

    GpuTimer timer;

    // Stream 1
    timer.Start();

    HANDLER_ERROR_ERR(hipMalloc((void**)&d_a.elements, ARRAY_BYTES));
    HANDLER_ERROR_ERR(hipMalloc((void**)&d_b.elements, ARRAY_BYTES));

    for (int i = 0; i < FULL_DATA_SIZE; i += N) {
        // copy the locked memory to the device, async
        HANDLER_ERROR_ERR(hipMemcpyAsync(d_a.elements, h_a.elements + i,
                                          ARRAY_BYTES, hipMemcpyHostToDevice,
                                          stream1));

        // copy the locked memory to the device, async
        HANDLER_ERROR_ERR(hipMemcpyAsync(d_b.elements, h_b.elements + i,
                                          ARRAY_BYTES, hipMemcpyHostToDevice,
                                          stream2));

        functionKernel1<<<DIMGRID, DIMBLOCK, 0, stream1>>>(d_a, N);
        HANDLER_ERROR_MSG("kernel panic!!!");

        functionKernel2<<<DIMGRID, DIMBLOCK, 0, stream2>>>(d_b, N);
        HANDLER_ERROR_MSG("kernel panic!!!");

        // copy from the device to the locked memory, async
        HANDLER_ERROR_ERR(hipMemcpyAsync(h_a.elements + i, d_a.elements,
                                          ARRAY_BYTES, hipMemcpyDeviceToHost,
                                          stream1));

        // copy from the device to the locked memory, async
        HANDLER_ERROR_ERR(hipMemcpyAsync(h_b.elements + i, d_b.elements,
                                          ARRAY_BYTES, hipMemcpyDeviceToHost,
                                          stream2));
    }

    // synchronization
    HANDLER_ERROR_ERR(hipStreamSynchronize(stream1));
    HANDLER_ERROR_ERR(hipStreamSynchronize(stream2));

    timer.Stop();

    // print time
    printf("Total time %f ms\n", timer.Elapsed());

    // destroy stream
    HANDLER_ERROR_ERR(hipStreamDestroy(stream1));
    HANDLER_ERROR_ERR(hipStreamDestroy(stream2));

    // free device memory
    HANDLER_ERROR_ERR(hipFree(d_a.elements));
    HANDLER_ERROR_ERR(hipFree(d_b.elements));
}

void onHost() {
    Vector<float> h_a, h_b;
    h_a.length = FULL_DATA_SIZE;
    h_b.length = FULL_DATA_SIZE;

    // allocate host locked memory
    HANDLER_ERROR_ERR(hipHostAlloc((void**)&h_a.elements, FULL_ARRAY_BYTES,
                                    hipHostMallocDefault));
    HANDLER_ERROR_ERR(hipHostAlloc((void**)&h_b.elements, FULL_ARRAY_BYTES,
                                    hipHostMallocDefault));

    int i;
    for (i = 0; i < FULL_DATA_SIZE; i++) {
        h_a.setElement(i, 0.0);
        h_b.setElement(i, 0.0);
    }

    // call device configuration
    onDevice(h_a, h_b);

    printf("-: successful execution :-\n");

    // free host memory
    HANDLER_ERROR_ERR(hipHostFree(h_a.elements));
    HANDLER_ERROR_ERR(hipHostFree(h_b.elements));
}

void checkDeviceProps() {
    // properties validation
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLER_ERROR_ERR(hipGetDevice(&whichDevice));
    HANDLER_ERROR_ERR(hipGetDeviceProperties(&prop, whichDevice));
    if (!prop.deviceOverlap) {
        printf(
            "Device will not handle overlaps, so no speed up from streams\n");
    }
}

int main() {
    checkDeviceProps();
    onHost();
}
